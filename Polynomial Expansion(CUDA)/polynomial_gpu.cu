
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

__global__ void polynomial_expansion (float* poly, int degree,
               int n, float* array) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if( i < n )
  {
    float result = 0.0;
    float xtothepowerof = 1.0;
    for ( int x = 0; x < degree; ++x)
    {
      result += xtothepowerof * poly[x];
      xtothepowerof *= array[i];
    }
    array[i] = result;
  }
}

int main (int argc, char* argv[])
{
  if (argc < 3)
  {
     std::cerr<<"usage: "<<argv[0]<<" n degree"<<std::endl;
     return -1;
  }

  int n = atoi(argv[1]);
  int degree = atoi(argv[2]);
  int nbiter = 1;

  float* array = new float[n];
  float* poly = new float[degree+1];

    std::chrono::time_point<std::chrono::system_clock> begin_time, end_time;

  for (int i=0; i<n; ++i)
    array[i] = 1.;

  for (int i=0; i<degree+1; ++i)
    poly[i] = 1.;

  float *d_array, *d_poly;

  
  hipMalloc((void **)&d_array, n*sizeof(float));
  hipMalloc((void **)&d_poly, (degree+1)*sizeof(float));
 
  hipMemcpy(d_array, array, n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_poly, poly, n*sizeof(float), hipMemcpyHostToDevice);
  

  begin_time = std::chrono::system_clock::now();

  for (int iter = 0; iter<nbiter; ++iter)
  polynomial_expansion<<<((n+255)/256) +1 , 256>>>(d_poly, degree, n, d_array);
  hipMemcpy(array, d_array, n*sizeof(float), hipMemcpyDeviceToHost);


  hipDeviceSynchronize();

  end_time = std::chrono::system_clock::now();
  std::chrono::duration<double> totaltime = (end_time-begin_time)/nbiter;

  {
    bool correct = true;
    int ind;
    for (int i=0; i< n; ++i) {
      if (fabs(array[i]-(degree+1))>0.01) {
        correct = false;
    ind = i;
      }
    }
    if (!correct)
      std::cerr<<"Result is incorrect. In particular array["<<ind<<"] should be "<<degree+1<<" not "<< array[ind]<<std::endl;
  }

  std::cout<<n<<" "<<degree<<" "<<totaltime.count()<<std::endl;

  delete[] array;
  delete[] poly;
 
  hipFree(d_array);
  hipFree(d_poly);

  return 0;
}